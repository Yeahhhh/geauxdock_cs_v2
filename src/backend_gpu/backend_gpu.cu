
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <ctime>

#include <geauxdock.h>
#include <size.h>
#include <toggle.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <geauxdock.h>
#include <toggle.h>
#include <util_print.h>

#include <yeah/cuda/runtime/wrapper.h>
#include <yeah/cuda/runtime/event.h>
#include <yeah/cuda/runtime/info.h>
#include <yeah/cuda/kernel/util.h>
//#include <yeah/c/timing.h>
#include <yeah/measuring/counting/timer.hpp>




#define CUDAASSERT(condition, code) \
    if (!(condition)) printf("Assertion failure. Code %s%n\n", code)

#include <backend_gpu.h>
#include "kernel_cuda_l2_reduce_notemplate.cu"
#include "kernel_cuda_l2_util.cu"
#include "kernel_cuda_l1_initcurand.cu"
#include "kernel_cuda_l1_montecarlo.cu"



// GPU timer can't measure memory copy
// CPU timer can't measure mc kernel



static void
InitCurand (hiprandState *s)
{
    const int mydevice = 0;
    hipDeviceProp_t prop;
    CUDA_ERR (hipGetDeviceProperties (&prop, mydevice));
    const int BperG = prop.multiProcessorCount * BperMP; // blocks per GPU


    srand (time (0));
    InitCurand_d <<< BperG, TperB >>> (s, rand ());
}



static void
Dock (Complex *ch,
    Record *rh,
    Complex *cd,
    Record *rd,
    hiprandState *curandstate_d)
{
    const int steps_total = ch->mcpara.steps_total;
    const int steps_per_dump = ch->mcpara.steps_per_dump;
    const size_t record_sz = sizeof (Record) * ch->size.n_rep;


    const int mydevice = 0;
    hipDeviceProp_t prop;
    CUDA_ERR (hipGetDeviceProperties (&prop, mydevice));
    const int BperG = prop.multiProcessorCount * BperMP; // blocks per GPU



    yeah::measuring::counting::Timer e[11];

    //yeah::cuda::EventSD e[11];

    CUDA_ERR (hipMemcpyAsync (cd, ch, sizeof (Complex), hipMemcpyHostToDevice));

    //GetPrintCudaFuncArributes ((void (*)) MonteCarlo_d, "MonteCarlo_d");
    //GetPrintCudaFuncArributes2 ((void (*)) MonteCarlo_d, "MonteCarlo_d", BperG, TperB, 0);


    e[10].Start ();


    e[3].Start ();
    MonteCarlo_d <<< BperG, TperB >>> (cd, rd, 0, 1, curandstate_d);
    CUDA_LAST_ERR ();
    e[3].Stop ();



    for (int s1 = 0; s1 < steps_total; s1 += steps_per_dump) {
        printf ("\t%d / %d \n", s1, steps_total);

        e[4].Start ();
        MonteCarlo_d <<< BperG, TperB >>> (cd, rd, s1, steps_per_dump, curandstate_d);
        CUDA_LAST_ERR ();
        //hipDeviceSynchronize();

        //yeah::measuring::counting::Timer eeee;
        //eeee.Start ();
        CUDA_ERR (hipMemcpy (rh, rd, record_sz, hipMemcpyDeviceToHost));
        // eeee.Stop ();
        // printf ("launcher: time of memory copy D2H %f\n", eeee.Span());
        e[4].Stop ();
//#include <kernel_dump.cpp>
    }

    e[10].Stop ();


#include "kernel_print_performance.cpp"
PrintResult (ch, rh);
//PrintSummary (ch);

}






Docker::Docker()
{
    const int mydevice = 0;
    hipDeviceProp_t prop;
    CUDA_ERR (hipGetDeviceProperties (&prop, mydevice));
    const int BperG = prop.multiProcessorCount * BperMP; // blocks per GPU
    printf ("BperG = %d\n", BperG);




    printf ("Initilizing GPU\n");
    printf ("sizeof record = %f MB\n", (float) sizeof (Record) * MAX_REP / 1024 / 1024);


    CUDA_ERR (hipHostMalloc ((void **) &complex, sizeof (Complex)));
    CUDA_ERR (hipHostMalloc ((void **) &record, sizeof (Record) * MAX_REP));

    CUDA_ERR (hipMalloc ((void **) &cd, sizeof (Complex)));
    CUDA_ERR (hipMalloc ((void **) &rd, sizeof (Record) * MAX_REP));
    CUDA_ERR (hipMalloc ((void **) &sd, sizeof (hiprandState) * TperB * BperG));

    hipDeviceSetCacheConfig (hipFuncCachePreferShared);
    InitCurand (sd);
}



Docker::~Docker()
{
    CUDA_ERR (hipFree (cd));
    CUDA_ERR (hipFree (rd));
    CUDA_ERR (hipFree (sd));
    CUDA_ERR (hipHostFree (record));
    CUDA_ERR (hipHostFree (complex));

    hipDeviceReset ();
}



void Docker::run()
{
    Dock (complex, record, cd, rd, sd);
}


